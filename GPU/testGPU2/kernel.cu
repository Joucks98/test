#include "hip/hip_runtime.h"
#include <stdio.h>
#define N 102400000*9
#define TPB 512
const int blockSize = N / TPB;
__device__ float scale(int i, int n)
{
    return ((float)i) / (n - 1);
}

__device__ float distance(float x1, float x2)
{
    return sqrt((x2 - x1)*(x2 - x1));
}

__global__ void distanceKernel(float *d_out, float ref, int len)
{
    const int i = blockIdx.x*blockDim.x + threadIdx.x;
    const float x = scale(i, len);
    d_out[i] = distance(x, ref);
    printf("i = %2d: dist from %f to %f is %f.\n", i, ref, x, d_out[i]);
}


void printDeviceProp(const hipDeviceProp_t &prop)
{
    printf("Device Name : %s.\n", prop.name);
    printf("totalGlobalMem : %d.\n", prop.totalGlobalMem);
    printf("sharedMemPerBlock : %d.\n", prop.sharedMemPerBlock);
    printf("regsPerBlock : %d.\n", prop.regsPerBlock);
    printf("warpSize : %d.\n", prop.warpSize);
    printf("memPitch : %d.\n", prop.memPitch);
    printf("maxThreadsPerBlock : %d.\n", prop.maxThreadsPerBlock);
    printf("maxThreadsDim[0 - 2] : %d %d %d.\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
    printf("maxGridSize[0 - 2] : %d %d %d.\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    printf("totalConstMem : %d.\n", prop.totalConstMem);
    printf("major.minor : %d.%d.\n", prop.major, prop.minor);
    printf("clockRate : %d.\n", prop.clockRate);
    printf("textureAlignment : %d.\n", prop.textureAlignment);
    printf("deviceOverlap : %d.\n", prop.deviceOverlap);
    printf("multiProcessorCount : %d.\n", prop.multiProcessorCount);
}

//CUDA ��ʼ��
bool InitCUDA()
{
    int count;

    //ȡ��֧��Cuda��װ�õ���Ŀ
    hipGetDeviceCount(&count);

    if (count == 0)
    {
        fprintf(stderr, "There is no device.\n");

        return false;
    }
    int i;
    for (i = 0; i < count; ++i)
    {

        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        //��ӡ�豸��Ϣ
        printDeviceProp(prop);

        if (hipGetDeviceProperties(&prop, i) == hipSuccess)
        {
            if (prop.major >= 1)
            {
                break;
            }
        }
    }

    if (i == count)
    {
        fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
        return false;
    }

    hipSetDevice(i);

    return true;

}


int main()
{
    if (!InitCUDA()) return 0;
    const float ref = 0.5f;

    // Declare a pointer for an array of floats
    float *d_out = 0;

    // Allocate device memory to store the output array
    hipMalloc(&d_out, N * sizeof(float));

    // Launch kernel to compute and store distance values
    distanceKernel << <N / TPB, TPB >> >(d_out, ref, N);

    hipFree(d_out); // Free the memory
    return 0;
}